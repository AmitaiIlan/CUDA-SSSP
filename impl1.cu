#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>

#include "cuda_error_check.cuh"
#include "utils.cuh"

__device__ int min_kernel(int a, int b){
        return a < b? a : b;
}

__global__ void swap_kernel(int* a, int* b, int n){
        const int tid = threadIdx.x + blockDim.x*blockIdx.x;
        const int nThreads = blockDim.x*gridDim.x;
        const int iter = n%nThreads == 0? n/nThreads : n/nThreads+1;

        for(int i = 0; i < iter; i++){
                int id = tid + i*nThreads;
                if(id < n){
                        int temp = a[id];
                        a[id] = b[id];
                        b[id] = temp;
                }
        }
}

__global__ void impl1_incore_kernel(edge* edges, int nEdges, int* distance, int* anyChange){
	const int idx = blockDim.x*blockIdx.x + threadIdx.x;
	const int nThreads = blockDim.x * gridDim.x;
	const int nWarps = nThreads%32 == 0? nThreads/32 : nThreads/32+1; //number of warps
	const int lane = idx & 31; //thread offset within a warp
	const int warpid = idx >> 5;

	int load = nEdges%nWarps == 0? nEdges/nWarps : nEdges/nWarps+1;
	int beg = load*warpid;
	int end = min_kernel(nEdges,beg+load);
	beg = beg+lane;

	for(int i = beg; i < end; i++){
		int u = edges[i].src;
		int v = edges[i].dest;
		int w = edges[i].w;
		int temp_dist = distance[u]+w;
		if(distance[u] == INF) continue;
		if(temp_dist < distance[v]){
			atomicMin(&distance[v], temp_dist);
			*anyChange = 1;
		}
	}
}

__global__ void impl1_outcore_kernel(edge* edges, int nEdges, int* distance_cur, int* distance_prev, int* anyChange){
	const int idx = blockDim.x*blockIdx.x + threadIdx.x;
	const int nThreads = blockDim.x * gridDim.x;
	const int nWarps = nThreads%32 == 0? nThreads/32 : nThreads/32+1; //number of warps
	const int lane = idx & 31; //thread offset within a warp
	const int warpid = idx >> 5;

	int load = nEdges%nWarps == 0? nEdges/nWarps : nEdges/nWarps+1;
	int beg = load*warpid;
	int end = min_kernel(nEdges,beg+load);
	beg = beg+lane;

	for(int i = beg; i < end; i++){
		int u = edges[i].src;
		int v = edges[i].dest;
		int w = edges[i].w;
		if(distance_prev[u] == INF) continue;
		if(distance_prev[u]+w < distance_cur[v]){
			atomicMin(&distance_cur[v], distance_prev[u]+w);
			*anyChange = 1;
		}
	}
}

void impl1_incore(int* results, edge* h_edges, int nEdges, int n, int blockSize, int blockNum){
	int nb = n*sizeof(int);
	int* d_anyChange = NULL;
	edge* d_edges = NULL;
	int* d_distance = NULL;
	hipMalloc((void**)&d_edges,nEdges*sizeof(edge));
	hipMalloc((void**)&d_distance,nb);
	hipMalloc((void**)&d_anyChange,sizeof(int));
	hipMemcpy(d_edges,h_edges,nEdges*sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(d_distance,results,nb,hipMemcpyHostToDevice);

	int nIter = 0;
    	setTime();
	for(int i = 0; i < n-1; i++){
		nIter++;
		hipMemset(d_anyChange, 0,sizeof(int));
		impl1_incore_kernel<<<blockNum,blockSize>>>(d_edges,nEdges,d_distance,d_anyChange);
	
		//break from loop if no changes
		int anyChange = 0;
		hipMemcpy(&anyChange,d_anyChange, sizeof(int),hipMemcpyDeviceToHost);
		if(!anyChange) break;
	}
	cout << "Time: " << getTime() << "ms\n";
	cout << "Iterations: " << nIter << "\n";
	
	hipMemcpy(results,d_distance,nb,hipMemcpyDeviceToHost);
	
	hipFree(d_edges);
	hipFree(d_distance);
	hipFree(d_anyChange);
}

void impl1_outcore(int* distance, edge* h_edges, int nEdges, int n, int blockSize, int blockNum){
	int nb = n*sizeof(int);
	int* d_anyChange = NULL;
	edge* d_edges = NULL;
	int* d_distance_cur = NULL;
	int* d_distance_prev = NULL;
	hipMalloc((void**)&d_edges,nEdges*sizeof(edge));
	hipMalloc((void**)&d_distance_cur,nb);
	hipMalloc((void**)&d_distance_prev,nb);
	hipMalloc((void**)&d_anyChange,sizeof(int));
	hipMemcpy(d_edges,h_edges,nEdges*sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(d_distance_cur,distance,nb,hipMemcpyHostToDevice);
	hipMemcpy(d_distance_prev,distance,nb,hipMemcpyHostToDevice);

	int nIter = 0;
    	setTime();
	for(int i = 0; i < n-1; i++){
		nIter++;
		hipMemset(d_anyChange, 0,sizeof(int));
		impl1_outcore_kernel<<<blockNum,blockSize>>>(d_edges,nEdges,d_distance_cur,d_distance_prev,d_anyChange);
	
		//break from loop if no changes
		int anyChange = 0;
		hipMemcpy(&anyChange,d_anyChange, sizeof(int),hipMemcpyDeviceToHost);
		if(!anyChange) break;

		swap_kernel<<<blockNum,blockSize>>>(d_distance_cur,d_distance_prev,n);
	}
	cout << "Time: " << getTime() << "ms\n";
	cout << "Iterations: " << nIter << "\n";
	
	hipMemcpy(distance,d_distance_cur,nb,hipMemcpyDeviceToHost);
	
	hipFree(d_edges);
	hipFree(d_distance_prev);
	hipFree(d_distance_cur);
	hipFree(d_anyChange);
}
