#include <cstring>
#include <stdexcept>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <vector>

#include "cuda_error_check.cuh"
#include "utils.cuh"

#include "impl2.cu"
#include "impl1.cu"

enum ProcessingType {BMF, TPE, Unknown_processing};
enum ProcessingType processingMethod;
enum SyncMode {InCore, OutCore};
enum SyncMode syncMethod;
enum SmemMode {UseSmem, UseNoSmem};
enum SmemMode smemMethod;
enum SortMode {SortDest,SortSrc};
enum SortMode sortMethod;

// Open files safely.
template <typename T_file>
void openFileToAccess( T_file& input_file, std::string file_name ) {
	input_file.open( file_name.c_str() );
	if( !input_file )
		throw std::runtime_error( "Failed to open specified file: " + file_name + "\n" );
}


// Execution entry point.
int main( int argc, char** argv )
{

	std::string usage =
		"\tCommand line arguments:\n\
			Input file: E.g., --input in.txt\n\
                        Output path: E.g., --output out.txt\n\
                        Block size: E.g., --bsize 1024\n\
                        Block count: E.g., --bcount 2\n\
			Processing method: E.g., --method bmf (bellman-ford), or tpe (to-process-edge)\n\
			Shared memory usage: E.g., --usesmem yes, or no \n\
			Sync method: E.g., --sync incore, or outcore\n\
			Edge Sorting: E.g., --sort dest (by destination), or src (by source)\n";

	try {

		ifstream inputFile;
		ofstream outputFile;
		string inputFileName;
		int selectedDevice = 0;
		hipDeviceProp_t props;
		int bsize = 0, bcount = 0;
		long long arbparam = 0;
		bool nonDirectedGraph = false;		// By default, the graph is directed.
		processingMethod = Unknown_processing;
		syncMethod = OutCore;
		smemMethod = UseNoSmem;
		sortMethod = SortDest;


		/********************************
		 * GETTING INPUT PARAMETERS.
		 ********************************/

		for( int iii = 1; iii < argc; ++iii )
			if ( !strcmp(argv[iii], "--method") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "bmf") )
				        processingMethod = BMF;
				else if ( !strcmp(argv[iii+1], "tpe") )
    				        processingMethod = TPE;
				else{
					std::cerr << "\n Un-recognized method parameter value \n\n";
					return( EXIT_FAILURE );
				}   
			}
			else if ( !strcmp(argv[iii], "--sync") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "incore") )
				        syncMethod = InCore;
				else if ( !strcmp(argv[iii+1], "outcore") )
    				        syncMethod = OutCore;
				else{
					std::cerr << "\n Un-recognized sync parameter value \n\n";
					return( EXIT_FAILURE );
				}  

			}
			else if ( !strcmp(argv[iii], "--usesmem") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "yes") )
				        smemMethod = UseSmem;
				else if ( !strcmp(argv[iii+1], "no") )
    				        smemMethod = UseNoSmem;
        			else{
					std::cerr << "\n Un-recognized usesmem parameter value \n\n";
					return( EXIT_FAILURE );
				}  
			}
			else if( !strcmp(argv[iii], "--sort") && iii != argc-1 ) {
				if ( !strcmp(argv[iii+1], "dest") )
					sortMethod = SortDest;
				else if(!strcmp(argv[iii+1], "src") )
					sortMethod = SortSrc;
				else{
					std::cerr << "\n Un-recognized sort parameter value \n\n";
					return( EXIT_FAILURE );
				}
			}
			else if( !strcmp( argv[iii], "--input" ) && iii != argc-1 /*is not the last one*/){
				inputFileName = std::string(argv[iii+1]);
				openFileToAccess< std::ifstream >( inputFile, inputFileName );
			}
			else if( !strcmp( argv[iii], "--output" ) && iii != argc-1 /*is not the last one*/)
				openFileToAccess< std::ofstream >( outputFile, std::string( argv[iii+1] ) );
			else if( !strcmp( argv[iii], "--bsize" ) && iii != argc-1 /*is not the last one*/)
				bsize = std::atoi( argv[iii+1] );
			else if( !strcmp( argv[iii], "--bcount" ) && iii != argc-1 /*is not the last one*/)
				bcount = std::atoi( argv[iii+1] );

		if(bsize <= 0 || bcount <= 0){
			std::cerr << "Usage: " << usage;
			throw std::runtime_error("\nAn initialization error happened.\nExiting.");
		}
		if( !inputFile.is_open() || processingMethod == Unknown_processing ) {
			std::cerr << "Usage: " << usage;
			throw std::runtime_error( "\nAn initialization error happened.\nExiting." );
		}
		if(smemMethod == UseSmem && processingMethod != BMF){
                        cerr << "Shared Memory is only supported for the \"bmf\" method\n";
			cerr << "Try using --method bmf\n";
                        throw std::runtime_error("An initialization error happened.\nExiting.");
                }
                if(smemMethod == UseSmem && syncMethod == InCore){
                        cerr << "Shared Memory is not supported for in-core sync method\n";
			cerr << "Try using --sync outcore\n";
                        throw std::runtime_error("An initialization error happened.\nExiting.");
                }
		if( !outputFile.is_open() )
			openFileToAccess< std::ofstream >( outputFile, "out.txt" );
		CUDAErrorCheck( hipSetDevice( selectedDevice ) );
        	hipGetDeviceProperties(&props, selectedDevice);
		char* deviceName = props.name;
		//cout << "Selected device ID: " << selectedDevice << ", device name: " << deviceName << endl;

		/********************************
		 * Read the input graph file.
		 ********************************/

		std::vector<initial_vertex> parsedGraph( 0 );
		uint nEdges = parse_graph(
				inputFile,		// Input file.
				parsedGraph,	// The parsed graph.
				arbparam,
				nonDirectedGraph );		// Arbitrary user-provided parameter.
		int nNodes = parsedGraph.size();
		cout << "Input graph: " << inputFileName << ", nodes: " << nNodes << ", edges: " << nEdges << endl;

		edge* edges = new edge[nEdges];
		sort_by_dest(edges,nEdges,&parsedGraph);
		int* results = new int[nNodes];
		std::fill_n(results,nNodes,INF);
		results[0] = 0;
		if(sortMethod == SortSrc){
			sort_by_src(edges,nEdges);
		}
		bool useShmem = smemMethod == UseSmem;

		cout << "Configurations: ";
		cout << (processingMethod==BMF? "bmf" : "tpe") << " ";
		cout << (syncMethod==InCore? "in-core" : "out-core") << " implementation, ";
		cout << "sorting by " << (sortMethod==SortSrc? "src" : "dest") << ", ";
		cout << (useShmem? "using shmem" : "no shmem") << ", ";
		cout << bcount << " blocks, " << bsize << " threads each.\n";

		/********************************
		 * Process the graph.
		 ********************************/
		
		switch(processingMethod){
		case BMF:
			if(syncMethod == InCore)
				impl1_incore(results,edges,nEdges,nNodes, bsize, bcount, deviceName);
			else
				impl1_outcore(results,edges,nEdges,nNodes, bsize, bcount, useShmem, deviceName);
			break;
		case TPE:
			if(syncMethod == InCore)
				impl2_incore(results,edges,nEdges,nNodes, bsize, bcount,deviceName);
			else
				impl2_outcore(results,edges,nEdges,nNodes, bsize, bcount,deviceName);
			break;
		default:
		    break;
		}

		testCorrectness(edges, results,nEdges, nNodes);
		saveResults(outputFile,results,nNodes);
		delete[] edges, results;

		/********************************
		 * It's done here.
		 ********************************/

		CUDAErrorCheck( hipDeviceReset() );
		return( EXIT_SUCCESS );

	}
	catch( const std::exception& strException ) {
		std::cerr << strException.what() << "\n";
		return( EXIT_FAILURE );
	}
	catch(...) {
		std::cerr << "An exception has occurred." << std::endl;
		return( EXIT_FAILURE );
	}

}
